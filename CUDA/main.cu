
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int * num1, int * num2, int * result){
    *result = *num1 + *num2;
}

int main(void){
    //host copies
    int     num1 = 4,
            num2 = 5,
            result;

    // device copies (GPU)
    int *p_num1, *p_num2, *p_result;

    // allocate memory on device (GPU)
    if (hipMalloc(&p_num1, sizeof(int)) != hipSuccess) {
        printf("Device memory allocation failure!\n");
        return 1;
    }
    hipMalloc(&p_num2, sizeof(int));
    hipMalloc(&p_result, sizeof(int));

    hipMemcpy(p_num1, &num1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(p_num2, &num2, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<1, 1>>>(p_num1, p_num2, p_result);

    // куда, откуда, кол-во байтов, Device<-Host or Host<-Device
    hipMemcpy(&result, p_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d \n", result);

    hipFree(p_num1);
    hipFree(p_num2);
    hipFree(p_result);
}

/*
nvcc main.cu && ./a.out
*/
